#include "hip/hip_runtime.h"
#include "cuda_settings.h"

template <typename T>
void print_matrix(const int &m, const int &n, const T *A, const int &lda);

template <>
void print_matrix(const int &m, const int &n, const float *A, const int &lda)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%0.2f ", A[j * lda + i]);
        }
        printf("\n");
    }
}

template <>
void print_matrix(const int &m, const int &n, const double *A, const int &lda)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%0.2f ", A[j * lda + i]);
        }
        printf("\n");
    }
}

template <>
void print_matrix(const int &m, const int &n, const hipComplex *A, const int &lda)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%0.2f + %0.2fj ", A[j * lda + i].x, A[j * lda + i].y);
        }
        printf("\n");
    }
}

template <>
void print_matrix(const int &m, const int &n, const hipDoubleComplex *A, const int &lda)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%12.6f + %12.6fj ", A[j * lda + i].x, A[j * lda + i].y);
        }
        printf("\n");
    }
}

extern "C"
{
    /*
     *cuda matrix operations
     */

    hipblasOperation_t char_to_cublas_trans(char trans)
    {
        hipblasOperation_t cuTrans;
        switch (trans)
        {
        case 'n':
        case 'N':
            cuTrans = HIPBLAS_OP_N;
            break;
        case 't':
        case 'T':
            cuTrans = HIPBLAS_OP_T;
            break;
        case 'c':
        case 'C':
            cuTrans = HIPBLAS_OP_C;
            break;
        default:
            exit(-1);
        }
        return cuTrans;
    }
    void run_ddot_(int *_n, double *x, double *y, double *z)
    {
        int n = *_n;
        double *dx, *dy, *dz;
        hipblasHandle_t blasHandle;
        CUDA_CHECK(hipMalloc((void **)&dx, n * sizeof(double)));
        CUDA_CHECK(hipMalloc((void **)&dy, n * sizeof(double)));
        CUDA_CHECK(hipMalloc((void **)&dz, sizeof(double)));

        // cublas set vector api also works
        CUDA_CHECK(hipMemcpy(dx, x, n * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dy, y, n * sizeof(double), hipMemcpyHostToDevice));

        hipblasCreate(&blasHandle);
        hipblasDdot(blasHandle, n, dx, 1, dy, 1, dz);
        // cublas get vector api also works
        CUDA_CHECK(hipMemcpy(z, dz, sizeof(double), hipMemcpyDeviceToHost));
        hipFree(dx);
        hipFree(dy);
        hipFree(dz);
        hipblasDestroy(blasHandle);
    }

    void run_cublas_dgemv_(char *_trans, int *_m, int *_n, double *_alpha, double *A, int *_lda, double *x, int *_incx, double *_beta, double *y, int *_incy)
    {
        char trans = *_trans;
        int m = *_m, n = *_n, lda = *_lda, incx = *_incx, incy = *_incy;
        double alpha = *_alpha, beta = *_beta;
        double *dA, *dX, *dY;
        hipblasHandle_t blasHandle;
        hipStream_t stream = NULL;
        CUDA_CHECK(hipMallocAsync((void **)&dA, m * n * sizeof(double), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dX, n * sizeof(double), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dY, m * sizeof(double), stream));

        CUBLAS_CHECK(hipblasCreate(&blasHandle));
        CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUBLAS_CHECK(hipblasSetStream(blasHandle, stream));

        CUBLAS_CHECK(hipblasSetMatrixAsync(m, n, sizeof(double), A, m, dA, m, stream));
        CUBLAS_CHECK(hipblasSetVectorAsync(n, sizeof(double), x, 1, dX, 1, stream));
        CUBLAS_CHECK(hipblasDgemv(blasHandle, char_to_cublas_trans(trans), m, n, &alpha, dA, lda, dX, incx, &beta, dY, incy));
        CUBLAS_CHECK(hipblasGetVectorAsync(m, sizeof(double), dY, 1, y, 1, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipFreeAsync(dA, stream));
        CUDA_CHECK(hipFreeAsync(dX, stream));
        CUDA_CHECK(hipFreeAsync(dY, stream));
        CUBLAS_CHECK(hipblasDestroy(blasHandle));
        CUDA_CHECK(hipStreamDestroy(stream));
    }

    void run_cublas_zgemv_(char *_trans, int *_m, int *_n, hipDoubleComplex *alpha, hipDoubleComplex *A, int *_lda, hipDoubleComplex *x, int *_incx, hipDoubleComplex *beta, hipDoubleComplex *y, int *_incy)
    {
        char trans = *_trans;
        int m = *_m, n = *_n, lda = *_lda, incx = *_incx, incy = *_incy;
        hipDoubleComplex *dA = nullptr, *dX = nullptr, *dY = nullptr;
        hipblasHandle_t blasHandle;
        hipStream_t stream = NULL;
        CUDA_CHECK(hipMallocAsync((void **)&dA, m * n * sizeof(hipDoubleComplex), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dX, n * sizeof(hipDoubleComplex), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dY, m * sizeof(hipDoubleComplex), stream));

        CUBLAS_CHECK(hipblasCreate(&blasHandle));
        CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUBLAS_CHECK(hipblasSetStream(blasHandle, stream));
        CUBLAS_CHECK(hipblasSetMatrixAsync(m, n, sizeof(hipDoubleComplex), A, m, dA, m, stream));
        CUBLAS_CHECK(hipblasSetVectorAsync(n, sizeof(hipDoubleComplex), x, 1, dX, 1, stream));
        CUBLAS_CHECK(hipblasZgemv(blasHandle, char_to_cublas_trans(trans), m, n, (hipDoubleComplex *)alpha, dA, lda, dX, incx, (hipDoubleComplex *)beta, dY, incy));
        CUBLAS_CHECK(hipblasGetVectorAsync(m, sizeof(hipDoubleComplex), dY, 1, y, 1, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipFreeAsync(dA, stream));
        CUDA_CHECK(hipFreeAsync(dX, stream));
        CUDA_CHECK(hipFreeAsync(dY, stream));
        CUBLAS_CHECK(hipblasDestroy(blasHandle));
        CUDA_CHECK(hipStreamDestroy(stream));
    }

    void run_cublas_dgemm_(char *_transa, char *_transb, int *_m, int *_n, int *_k, double *_alpha, double *A, int *_lda, double *B, int *_ldb, double *_beta, double *C, int *_ldc)
    {
        char transa = *_transa, transb = *_transb;
        int m = *_m, n = *_n, k = *_k, lda = *_lda, ldb = *_ldb, ldc = *_ldc;
        double *dA = nullptr, *dB = nullptr, *dC = nullptr, alpha = *_alpha, beta = *_beta;
        hipblasHandle_t blasHandle;
        hipStream_t stream = NULL;
        CUDA_CHECK(hipMallocAsync((void **)&dA, m * k * sizeof(double), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dB, k * n * sizeof(double), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dC, m * n * sizeof(double), stream));

        CUBLAS_CHECK(hipblasCreate(&blasHandle));
        CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUBLAS_CHECK(hipblasSetStream(blasHandle, stream));
        CUBLAS_CHECK(hipblasSetMatrixAsync(m, k, sizeof(double), A, m, dA, m, stream));
        CUBLAS_CHECK(hipblasSetMatrixAsync(k, n, sizeof(double), B, k, dB, k, stream));
        CUBLAS_CHECK(hipblasDgemm(blasHandle, char_to_cublas_trans(transa), char_to_cublas_trans(transb), m, n, k, &alpha, dA, lda, dB, ldb, &beta, dC, ldc));
        CUBLAS_CHECK(hipblasGetMatrixAsync(m, n, sizeof(double), dC, m, C, m, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipFreeAsync(dA, stream));
        CUDA_CHECK(hipFreeAsync(dB, stream));
        CUDA_CHECK(hipFreeAsync(dC, stream));
        hipblasDestroy(blasHandle);
        CUDA_CHECK(hipStreamDestroy(stream));
    }

    void run_cublas_zgemm_(char *_transa, char *_transb, int *_m, int *_n, int *_k, hipDoubleComplex *alpha, hipDoubleComplex *A, int *_lda, hipDoubleComplex *B, int *_ldb, hipDoubleComplex *beta, hipDoubleComplex *C, int *_ldc)
    {
        char transa = *_transa, transb = *_transb;
        int m = *_m, n = *_n, k = *_k, lda = *_lda, ldb = *_ldb, ldc = *_ldc;
        hipDoubleComplex *dA = nullptr, *dB = nullptr, *dC = nullptr;
        hipblasHandle_t blasHandle;
        hipStream_t stream = NULL;

        CUDA_CHECK(hipMallocAsync((void **)&dA, m * k * sizeof(hipDoubleComplex), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dB, k * n * sizeof(hipDoubleComplex), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dC, m * n * sizeof(hipDoubleComplex), stream));

        CUBLAS_CHECK(hipblasCreate(&blasHandle));
        CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUBLAS_CHECK(hipblasSetStream(blasHandle, stream));
        CUBLAS_CHECK(hipblasSetMatrixAsync(m, k, sizeof(hipDoubleComplex), A, m, dA, m, stream));
        CUBLAS_CHECK(hipblasSetMatrixAsync(k, n, sizeof(hipDoubleComplex), B, k, dB, k, stream));
        CUBLAS_CHECK(hipblasZgemm(blasHandle, char_to_cublas_trans(transa), char_to_cublas_trans(transb), m, n, k, (hipDoubleComplex *)alpha, dA, lda, dB, ldb, (hipDoubleComplex *)beta, dC, ldc));
        CUBLAS_CHECK(hipblasGetMatrixAsync(m, n, sizeof(hipDoubleComplex), dC, m, C, m, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));

        CUDA_CHECK(hipFreeAsync(dA, stream));
        CUDA_CHECK(hipFreeAsync(dB, stream));
        CUDA_CHECK(hipFreeAsync(dC, stream));

        hipblasDestroy(blasHandle);
        CUDA_CHECK(hipStreamDestroy(stream));
    }

    void run_cublas_zgemm_strided_batched_(char *_transa, char *_transb, int *_m, int *_n, int *_k, hipDoubleComplex *alpha, hipDoubleComplex *A, int *_lda, long long int *_stridea, hipDoubleComplex *B, int *_ldb, long long int *_strideb, hipDoubleComplex *beta, hipDoubleComplex *C, int *_ldc, long long int *_stridec, int *_batch_count)
    {
        char transa = *_transa, transb = *_transb;
        int m = *_m, n = *_n, k = *_k, lda = *_lda, ldb = *_ldb, ldc = *_ldc, stridea = *_stridea, strideb = *_strideb, stridec = *_stridec, batchCount = *_batch_count;
        hipDoubleComplex *dA = nullptr, *dB = nullptr, *dC = nullptr;
        hipblasHandle_t blasHandle;
        hipStream_t stream = NULL;

        CUDA_CHECK(hipMallocAsync((void **)&dA, batchCount * m * k * sizeof(hipDoubleComplex), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dB, batchCount * k * n * sizeof(hipDoubleComplex), stream));
        CUDA_CHECK(hipMallocAsync((void **)&dC, batchCount * m * n * sizeof(hipDoubleComplex), stream));

        CUBLAS_CHECK(hipblasCreate(&blasHandle));
        CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUBLAS_CHECK(hipblasSetStream(blasHandle, stream));

        CUDA_CHECK(hipMemcpyAsync(dA, A, batchCount * m * k * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(dB, B, batchCount * k * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream));

        CUBLAS_CHECK(hipblasZgemmStridedBatched(blasHandle, char_to_cublas_trans(transa), char_to_cublas_trans(transb), m, n, k, (hipDoubleComplex *)alpha, dA, lda, stridea, dB, ldb, strideb, (hipDoubleComplex *)beta, dC, ldc, stridec, batchCount));

        CUDA_CHECK(hipMemcpyAsync(C, dC, batchCount * m * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));

        CUDA_CHECK(hipFreeAsync(dA, stream));
        CUDA_CHECK(hipFreeAsync(dB, stream));
        CUDA_CHECK(hipFreeAsync(dC, stream));

        hipblasDestroy(blasHandle);
        CUDA_CHECK(hipStreamDestroy(stream));
    }

    void run_cublas_zgemm_strided_batched_no_ab_(char *_transa, char *_transb, int *_m, int *_n, int *_k, hipDoubleComplex *A, int *_lda, long long int *_stridea, hipDoubleComplex *B, int *_ldb, long long int *_strideb, hipDoubleComplex *C, int *_ldc, long long int *_stridec, int *_batch_count)
    {
        char transa = *_transa, transb = *_transb;
        int m = *_m, n = *_n, k = *_k, lda = *_lda, ldb = *_ldb, ldc = *_ldc, stridea = *_stridea, strideb = *_strideb, stridec = *_stridec, batchCount = *_batch_count;
        hipDoubleComplex *dA = nullptr, *dB = nullptr, *dC = nullptr;
        hipblasHandle_t blasHandle;
        hipStream_t stream = NULL;

        CUDA_CHECK(hipMalloc((void **)&dA, batchCount * m * k * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMalloc((void **)&dB, batchCount * k * n * sizeof(hipDoubleComplex)));
        CUDA_CHECK(hipMalloc((void **)&dC, batchCount * m * n * sizeof(hipDoubleComplex)));

        CUBLAS_CHECK(hipblasCreate(&blasHandle));
        CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUBLAS_CHECK(hipblasSetStream(blasHandle, stream));

        CUDA_CHECK(hipMemcpyAsync(dA, A, batchCount * m * k * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(dB, B, batchCount * k * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream));

        CUBLAS_CHECK(hipblasZgemmStridedBatched(blasHandle, char_to_cublas_trans(transa), char_to_cublas_trans(transb), m, n, k, (hipDoubleComplex *)&cone_, dA, lda, stridea, dB, ldb, strideb, (hipDoubleComplex *)&czero_, dC, ldc, stridec, batchCount));

        CUDA_CHECK(hipMemcpyAsync(C, dC, batchCount * m * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));

        CUDA_CHECK(hipFree(dA));
        CUDA_CHECK(hipFree(dB));
        CUDA_CHECK(hipFree(dC));

        hipblasDestroy(blasHandle);
        CUDA_CHECK(hipStreamDestroy(stream));
    }

    /*
     *cuSolver
     */

    hipblasFillMode_t char_to_cublas_fillmode(char _uplo)
    {
        // Fortran follows columns while C/C++ follow rows
        hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
        switch (_uplo)
        {
        case 'u':
        case 'U':
            uplo = HIPBLAS_FILL_MODE_UPPER;
            break;
        case 'l':
        case 'L':
            uplo = HIPBLAS_FILL_MODE_LOWER;
        default:
            break;
        }
        return uplo;
    }
    void run_cusolver_zheevj_(int *_m, int *_n, char *_uplo, hipDoubleComplex *A, hipDoubleComplex *V, double *W)
    {
        const int m = *_m, n = *_n, lda = (m>n)?m:n;

        hipsolverHandle_t cusolverH = NULL;
        hipStream_t stream = NULL;
        hipsolverSyevjInfo_t syevj_params = NULL;
        hipDoubleComplex *d_A = nullptr, *d_work = nullptr;
        double *d_W = nullptr;
        int *devInfo = nullptr, lwork = 0, info_gpu = 0;

        /* configuration of syevj  */
        const double tol = 1.e-7;
        const int max_sweeps = 15;
        const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
        const hipblasFillMode_t uplo = char_to_cublas_fillmode(*_uplo);

        /* step 1: create cusolver handle, bind a stream */
        CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

        CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

        /* step 2: configuration of syevj */
        CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params));

        /* default value of tolerance is machine zero */
        CUSOLVER_CHECK(hipsolverDnXsyevjSetTolerance(syevj_params, tol));

        /* default value of max. sweeps is 100 */
        CUSOLVER_CHECK(hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps));

        /* step 3: copy A to device */
        CUDA_CHECK(hipMallocAsync(reinterpret_cast<void **>(&d_A), sizeof(hipDoubleComplex) * lda * m, stream));
        CUDA_CHECK(hipMallocAsync(reinterpret_cast<void **>(&d_W), sizeof(double) * m, stream));
        CUDA_CHECK(hipMallocAsync(reinterpret_cast<void **>(&devInfo), sizeof(int), stream));

        CUDA_CHECK(
            hipMemcpyAsync(d_A, A, sizeof(hipDoubleComplex) * lda * m, hipMemcpyHostToDevice, stream));

        /* step 4: query working space of syevj */
        CUSOLVER_CHECK(
            hipsolverDnZheevj_bufferSize(cusolverH, jobz, uplo, m, d_A, lda, d_W, &lwork, syevj_params));

        CUDA_CHECK(hipMallocAsync(reinterpret_cast<void **>(&d_work), sizeof(hipDoubleComplex) * lwork, stream));

        /* step 5: compute eigen-pair   */
        CUSOLVER_CHECK(hipsolverDnZheevj(cusolverH, jobz, uplo, m, d_A, lda, d_W, d_work, lwork, devInfo, syevj_params));

        CUDA_CHECK(hipMemcpyAsync(V, d_A, sizeof(hipDoubleComplex) * lda * m, hipMemcpyDeviceToHost, stream));
        CUDA_CHECK(hipMemcpyAsync(W, d_W, sizeof(double) * m, hipMemcpyDeviceToHost, stream));
        CUDA_CHECK(hipMemcpyAsync(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));

        if (0 > info_gpu)
        {
            printf("%d-th parameter is wrong \n", -info_gpu);
            exit(1);
        }
#ifdef DEBUG
        if (0 == info_gpu)
            printf("syevj converges \n");
        else
            printf("WARNING: info = %d : syevj does not converge \n", info_gpu);

        printf("Eigenvalue = (matlab base-1), ascending order\n");
        for (int i = 0; i < m; i++)
        {
            printf("W[%d] = %E\n", i + 1, W[i]);
        }
#endif

        /* free resources */
        CUDA_CHECK(hipFreeAsync(d_A, stream));
        CUDA_CHECK(hipFreeAsync(d_W, stream));
        CUDA_CHECK(hipFreeAsync(devInfo, stream));
        CUDA_CHECK(hipFreeAsync(d_work, stream));

        CUSOLVER_CHECK(hipsolverDnDestroySyevjInfo(syevj_params));

        CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

        CUDA_CHECK(hipStreamDestroy(stream));
    }
}